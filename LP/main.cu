#include <hip/hip_runtime.h>
#include ""
#include "hip/hip_runtime_api.h"
#include <glm/glm.hpp>

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include "FileIO.h"
#include "Auxilary.h"
#include "lp.h"


////////////////////////////////////
//Main

int main(int argc, const char* argv[])
{

	
	//------------------------------------------
	//handle args
	if (argc != 3) {
		printf("\nIncorrect Number of Arguments!\n");
		printf("Correct Usages/Syntax:\n");
		printf("Argument 1) File-Name -- Name of input file in benchmarks folder. Cannot contain spaces\n");
		printf("Argument 2) Batch-size -- Number of LPs to be solved\n");
		return 1;
	}

	// Run benchmark
	lplibBenchmark(argv[1], atoi(argv[2]));
	
  return 0;
}
